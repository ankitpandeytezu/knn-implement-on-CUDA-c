#include<hip/hip_runtime.h>
#include<stdio.h>

__global__ void checkIndex(void)
{
        printf("threadIdx: (%d %d %d) blockIdx: (%d %d %d) blockDim: (%d %d %d) "
                " gridDim: (%d %d %d)\n",threadIdx.x,threadIdx.y,threadIdx.z,blockIdx.x,blockIdx.y,blockIdx.z,blockDim.x,blockDim.y,                  blockDim.z,gridDim.x,gridDim.y,gridDim.z);  
}
  

int main(int args, char **argv)
{
    int n=6;
    // define grid and block structure//

    dim3 block(3);
    dim3 grid((n+block.x-1)/block.x);
    
// ckeck grid and block dimension from host side//
    printf("grid.x %d grid.y %d grid.x %d\n",grid.x,grid.y,grid.z);
    printf("block.x %d block.y %d block.z %d\n",block.x,block.y,block.z);
// check grid and block from device side//

    checkIndex <<<grid, block>>> ();
    hipDeviceReset();
    return 0;

}
