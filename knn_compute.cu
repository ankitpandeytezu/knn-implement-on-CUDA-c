#include "hip/hip_runtime.h"
/** Modifed version of knn-CUDA from https://github.com/vincentfpgarcia/kNN-CUDA
 * The modifications are
 *      removed texture memory usage
 *      removed split query KNN computation
 *      added feature extraction with bilinear interpolation
 *
 * Last modified by Christopher B. Choy <chrischoy@ai.stanford.edu> 12/23/2016
 */

// Includes
#include <cstdio>
#include<stdio.h>
#include "hip/hip_runtime.h"

// Constants used by the program
#define BLOCK_DIM                      16


//-----------------------------------------------------------------------------------------------//
//                                            KERNELS                                            //
//-----------------------------------------------------------------------------------------------//
__global__ void extract_with_interpolation(   int nthreads, float *data, float *n_xy_coords, float *extracted_data,
    int n_max_coord, int channels, int height, int width) 
   {

  int x0, x1, y0, y1, nc;
  float wx0, wx1, wy0, wy1;
  int n, nd;
  float x, y;

  for (int index = blockIdx.x * blockDim.x + threadIdx.x;
       index < (nthreads);
       index += blockDim.x * gridDim.x) {
    n = (index / n_max_coord);
    nd = n * n_max_coord * channels;
    x = n_xy_coords[index * 2];
    y = n_xy_coords[index * 2 + 1];

    x0 = static_cast<int>(floor(x));
    x1 = x0 + 1;
    y0 = static_cast<int>(floor(y));
    y1 = y0 + 1;

    x0 = x0 <= 0 ? 0 : (x0 >= (width - 1)  ? (width - 1) : x0);
    y0 = y0 <= 0 ? 0 : (y0 >= (height - 1) ? (height - 1) : y0);
    x1 = x1 <= 0 ? 0 : (x1 >= (width - 1)  ? (width - 1) : x1);
    y1 = y1 <= 0 ? 0 : (y1 >= (height - 1) ? (height - 1) : y1);

    wx0 = static_cast<float>(x1) - x;
    wx1 = x - x0;
    wy0 = static_cast<float>(y1) - y;
    wy1 = y - y0;

    if(x0 == x1){ wx0 = 1; wx1 = 0; }
    if(y0 == y1){ wy0 = 1; wy1 = 0; }
    for(int c=0; c < channels; c++) {
      nc = (n * channels + c) * height;
      // extracted_data[index * channels + c] = wy0 * wx0 * data[(nc + y0) * width + x0]
      // extracted_data[nd + index % n_max_coord + n_max_coord * c] = index;
      extracted_data[nd + index % n_max_coord + n_max_coord * c] = wy0 * wx0 * data[(nc + y0) * width + x0]
       + wy1 * wx0 * data[(nc + y1) * width + x0]
       + wy0 * wx1 * data[(nc + y0) * width + x1]
       + wy1 * wx1 * data[(nc + y1) * width + x1];
    }
  }
}

/**
  * Computes the distance between two matrix A (reference points) and
  * B (query points) containing respectively wA and wB points.
  *
  * @param A     pointer on the matrix A
  * @param wA    width of the matrix A = number of points in A
  * @param B     pointer on the matrix B
  * @param wB    width of the matrix B = number of points in B
  * @param dim   dimension of points = height of matrices A and B
  * @param AB    pointer on the matrix containing the wA*wB distances computed
  */
__global__ void cuComputeDistanceGlobal( float* A, int wA,
    float* B, int wB, int dim,  float* AB){

  // Declaration of the shared memory arrays As and Bs used to store the sub-matrix of A and B
  __shared__ float shared_A[BLOCK_DIM][BLOCK_DIM];
  __shared__ float shared_B[BLOCK_DIM][BLOCK_DIM];

  // Sub-matrix of A (begin, step, end) and Sub-matrix of B (begin, step)
  __shared__ int begin_A;
  __shared__ int begin_B;
  __shared__ int step_A;
  __shared__ int step_B;
  __shared__ int end_A;

  // Thread index
  int tx = threadIdx.x;
  int ty = threadIdx.y;

  // Other variables
  float tmp;
  float ssd = 0;

  // Loop parameters
  begin_A = BLOCK_DIM * blockIdx.y;
  begin_B = BLOCK_DIM * blockIdx.x;
  step_A  = BLOCK_DIM * wA;
  step_B  = BLOCK_DIM * wB;
  end_A   = begin_A + (dim-1) * wA;

    // Conditions
  int cond0 = (begin_A + tx < wA); // used to write in shared memory
  int cond1 = (begin_B + tx < wB); // used to write in shared memory & to computations and to write in output matrix
  int cond2 = (begin_A + ty < wA); // used to computations and to write in output matrix

  // Loop over all the sub-matrices of A and B required to compute the block sub-matrix
  for (int a = begin_A, b = begin_B; a <= end_A; a += step_A, b += step_B) {
    // Load the matrices from device memory to shared memory; each thread loads one element of each matrix
    if (a/wA + ty < dim){
      shared_A[ty][tx] = (cond0)? A[a + wA * ty + tx] : 0;
      shared_B[ty][tx] = (cond1)? B[b + wB * ty + tx] : 0;
    }
    else{
      shared_A[ty][tx] = 0;
      shared_B[ty][tx] = 0;
    }

    // Synchronize to make sure the matrices are loaded
    __syncthreads();

    // Compute the difference between the two matrixes; each thread computes one element of the block sub-matrix
    if (cond2 && cond1){
      for (int k = 0; k < BLOCK_DIM; ++k){
        tmp = shared_A[k][ty] - shared_B[k][tx];
        ssd += tmp*tmp;
      }
    }

    // Synchronize to make sure that the preceding computation is done before loading two new sub-matrices of A and B in the next iteration
    __syncthreads();
  }

  // Write the block sub-matrix to device memory; each thread writes one element
  if (cond2 && cond1)
    AB[(begin_A + ty) * wB + begin_B + tx] = ssd;
}


/**
  * Gathers k-th smallest distances for each column of the distance matrix in the top.
  *
  * @param dist        distance matrix
  * @param ind         index matrix
  * @param width       width of the distance matrix and of the index matrix
  * @param height      height of the distance matrix and of the index matrix
  * @param k           number of neighbors to consider
  */
__global__ void cuInsertionSort(float *dist, int *ind, int width, int height, int k){

  // Variables
  int l, i, j;
  float *p_dist;
  int   *p_ind;
  float curr_dist, max_dist;
  int   curr_row,  max_row;
  unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;

  if (xIndex<width){
    // Pointer shift, initialization, and max value
    p_dist   = dist + xIndex;
    p_ind    = ind  + xIndex;
    max_dist = p_dist[0];
    p_ind[0] = 1;

    // Part 1 : sort kth firt elementZ
    for (l=1; l<k; l++){
      curr_row  = l * width;
      curr_dist = p_dist[curr_row];
      if (curr_dist<max_dist){
        i=l-1;
        for (int a=0; a<l-1; a++){
          if (p_dist[a*width]>curr_dist){
            i=a;
            break;
          }
        }
        for (j=l; j>i; j--){
          p_dist[j*width] = p_dist[(j-1)*width];
          p_ind[j*width]   = p_ind[(j-1)*width];
        }
        p_dist[i*width] = curr_dist;
        p_ind[i*width]   = l+1;
      } else {
        p_ind[l*width] = l+1;
      }
      max_dist = p_dist[curr_row];
    }

    // Part 2 : insert element in the k-th first lines
    max_row = (k-1)*width;
    for (l=k; l<height; l++){
      curr_dist = p_dist[l*width];
      if (curr_dist<max_dist){
        i=k-1;
        for (int a=0; a<k-1; a++){
          if (p_dist[a*width]>curr_dist){
            i=a;
            break;
          }
        }
        for (j=k-1; j>i; j--){
          p_dist[j*width] = p_dist[(j-1)*width];
          p_ind[j*width]   = p_ind[(j-1)*width];
        }
        p_dist[i*width] = curr_dist;
        p_ind[i*width]   = l+1;
        max_dist             = p_dist[max_row];
      }
    }
  }
}


/**
  * Computes the square root of the first line (width-th first element)
  * of the distance matrix.
  *
  * @param dist    distance matrix
  * @param width   width of the distance matrix
  * @param k       number of neighbors to consider
  */
__global__ void cuParallelSqrt(float *dist, int width, int k){
    unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int yIndex = blockIdx.y * blockDim.y + threadIdx.y;
  if (xIndex<width && yIndex<k)
    dist[yIndex*width + xIndex] = sqrt(dist[yIndex*width + xIndex]);
}


//-----------------------------------------------------------------------------------------------//
//                                   K-th NEAREST NEIGHBORS                                      //
//-----------------------------------------------------------------------------------------------//


/**
  * Prints the error message return during the memory allocation.
  *
  * @param error        error value return by the memory allocation function
  * @param memorySize   size of memory tried to be allocated
  */
void printErrorMessage(hipError_t error, int memorySize){
  printf("==================================================\n");
  printf("MEMORY ALLOCATION ERROR  : %s\n", hipGetErrorString(error));
  printf("Whished allocated memory : %d\n", memorySize);
  printf("==================================================\n");
}


/**
  * Feature extraction algorithm
  * - Initialize CUDA
  * - Allocate device memory
  * - Copy data (activation, coordinates) from host to device memory
  * - Extract features from the coordinates using bilinear interpolation
  * - Copy extracted features from device to host memory
  *
  * @param activation            reference feature map
  * @param n_batch               number of feature maps
  * @param n_channel             size of the feature dimension
  * @param height                height of the feature map
  * @param width                 width of the feature map
  * @param coords                coordinates of the points for extraction
  * @param extracted_activation  pointer for the final extracted features
  *
  */
void extract_cuda(float* activation, int n_batch, int n_channel, int height,
    int width, float* coords, int n_max_coord, int dim_coord,
    float *extracted_activation){
  // activation n_batch x n_channel x height x width
  // coords n_batch x n_max_coord x dim_coord
  // uninitialized empty pointer which will be filled with extracted_activation
  // n_batch x n_channel x n_max_coord. KNN requires dim x n_feature format
  unsigned int size_of_float = sizeof(float);

  // Variables
  float *activation_device;
  float *coord_device;
  float *extracted_activation_device;

  // CUDA Initialisation
  hipInit(0);

  // Allocation of global memory for query points and for distances, CUDA_CHECK
  hipMalloc((void **) &activation_device,
      n_batch * n_channel * height * width * size_of_float);
  hipMalloc((void **) &extracted_activation_device,
      n_batch * n_channel * n_max_coord * size_of_float);
  hipMalloc((void **) &coord_device,
      n_batch * n_max_coord * dim_coord * size_of_float);

  // Grids ans threads
  dim3 g_size_r((n_batch * n_max_coord * dim_coord) / 256, 1, 1);
  dim3 t_size_r(256, 1, 1);
  if ((n_batch * n_max_coord * dim_coord) % 256 != 0) g_size_r.x += 1;

  hipMemset(extracted_activation_device, 0, n_batch * n_channel * n_max_coord * size_of_float);

  // Copy coordinates to the device
  hipMemcpy(coord_device, &coords[0],
     n_batch * n_max_coord * dim_coord * size_of_float,
     hipMemcpyHostToDevice);

  // Copy of part of query actually being treated
  hipMemcpy(activation_device, &activation[0],
      n_batch * n_channel * height * width * size_of_float,
      hipMemcpyHostToDevice);

  // Grids ans threads
  dim3 g_size((n_batch * n_max_coord) / 256, 1, 1);
  dim3 t_size(256, 1, 1);
  if ((n_batch * n_max_coord) % 256 != 0) g_size.x += 1;

  extract_with_interpolation<<<g_size, t_size>>>(n_batch * n_max_coord,
    activation_device, coord_device, extracted_activation_device,
    n_max_coord, n_channel, height, width);

  // Memory copy of output from device to host
  hipMemcpy(extracted_activation, &extracted_activation_device[0],
      n_batch * n_channel * n_max_coord * size_of_float,
      hipMemcpyDeviceToHost);

  // Free memory
  hipFree(coord_device);
  hipFree(activation_device);
  hipFree(extracted_activation_device);
}

/**
  * K nearest neighbor algorithm
  * - Initialize CUDA
  * - Allocate device memory
  * - Copy point sets (reference and query points) from host to device memory
  * - Compute the distances + indexes to the k nearest neighbors for each query point
  * - Copy distances from device to host memory
  *
  * @param ref_host      reference points ; pointer to linear matrix
  * @param ref_width     number of reference points ; width of the matrix
  * @param query_host    query points ; pointer to linear matrix
  * @param query_width   number of query points ; width of the matrix
  * @param height        dimension of points ; height of the matrices
  * @param k             number of neighbor to consider
  * @param dist_host     distances to k nearest neighbors ; pointer to linear matrix
  * @param dist_host     indexes of the k nearest neighbors ; pointer to linear matrix
  *
  */
void knn_cuda(float* ref_host, int ref_width, float* query_host, int query_width, int height, int k, float* dist_host, int* ind_host)
  {

  unsigned int size_of_float = sizeof(float);
  unsigned int size_of_int   = sizeof(int);

  // Variables
  float  *query_dev;
  float  *ref_dev;
  float  *dist_dev;
  int    *ind_dev;

  // CUDA Initialisation
  hipInit(0);

  // Allocation of global memory for query points and for distances, CUDA_CHECK
  hipMalloc((void **) &query_dev, query_width * height * size_of_float);
  hipMalloc((void **) &dist_dev, query_width * ref_width * size_of_float);

  // Allocation of global memory for indexes CUDA_CHECK
  hipMalloc((void **) &ind_dev, query_width * k * size_of_int);

  // Allocation of global memory CUDA_CHECK
  hipMalloc( (void **) &ref_dev, ref_width * height * size_of_float);

  hipMemcpy(ref_dev, &ref_host[0], ref_width * height * size_of_float,
      hipMemcpyHostToDevice);

  // Copy of part of query actually being treated
  hipMemcpy(query_dev, &query_host[0],
      query_width * height * size_of_float, hipMemcpyHostToDevice);

  // Grids ans threads
  dim3 g_16x16(query_width/16, ref_width/16, 1);
  dim3 t_16x16(16, 16, 1);
  if (query_width%16 != 0) g_16x16.x += 1;
  if (ref_width  %16 != 0) g_16x16.y += 1;
  //
  dim3 g_256x1(query_width/256, 1, 1);
  dim3 t_256x1(256, 1, 1);
  if (query_width%256 != 0) g_256x1.x += 1;

  dim3 g_k_16x16(query_width/16, k/16, 1);
  dim3 t_k_16x16(16, 16, 1);
  if (query_width%16 != 0) g_k_16x16.x += 1;
  if (k  %16 != 0) g_k_16x16.y += 1;

  // Kernel 1: Compute all the distances
  cuComputeDistanceGlobal<<<g_16x16,t_16x16>>>(ref_dev, ref_width,
      query_dev, query_width, height, dist_dev);

  // Kernel 2: Sort each column
  cuInsertionSort<<<g_256x1,t_256x1>>>(dist_dev, ind_dev,
      query_width, ref_width, k);

  // Kernel 3: Compute square root of k first elements
  cuParallelSqrt<<<g_k_16x16,t_k_16x16>>>(dist_dev, query_width, k);

  // Memory copy of output from device to host
  hipMemcpy(&dist_host[0], dist_dev,
      query_width * k *size_of_float, hipMemcpyDeviceToHost);

  hipMemcpy(&ind_host[0], ind_dev,
      query_width * k * size_of_int, hipMemcpyDeviceToHost);

  // Free memory
  hipFree(ref_dev);
  hipFree(ind_dev);
  hipFree(query_dev);
  hipFree(dist_dev);
}


/**
  * Example of use of kNN search CUDA.
  */
int main(void){
  // Variables and parameters
  float* ref;                 // Pointer to reference point array
  float* query;               // Pointer to query point array
  float* dist;                // Pointer to distance array
  int*   ind;                 // Pointer to index array
  int    ref_nb     = 4096;   // Reference point number, max=65535
  int    query_nb   = 4096;   // Query point number,     max=65535
  int    dim        = 32;     // Dimension of points
  int    k          = 20;     // Nearest neighbors to consider
  int    iterations = 100;
  int    i;

  // Memory allocation
  ref    = (float *) malloc(ref_nb   * dim * sizeof(float));
  query  = (float *) malloc(query_nb * dim * sizeof(float));
  dist   = (float *) malloc(query_nb * k * sizeof(float));
  ind    = (int *)   malloc(query_nb * k * sizeof(float));

  // Init 
  srand(time(NULL));
  for (i=0 ; i<ref_nb   * dim ; i++) ref[i]    = (float)rand() / (float)RAND_MAX;
  for (i=0 ; i<query_nb * dim ; i++) query[i]  = (float)rand() / (float)RAND_MAX;

  // Variables for duration evaluation
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float elapsed_time;

  // Display informations
  printf("Number of reference points      : %6d\n", ref_nb  );
  printf("Number of query points          : %6d\n", query_nb);
  printf("Dimension of points             : %4d\n", dim     );
  printf("Number of neighbors to consider : %4d\n", k       );
  printf("Processing kNN search           :"                );

  // Call kNN search CUDA
  hipEventRecord(start, 0);
  for (i=0; i<iterations; i++) {
    knn_cuda(ref, ref_nb, query, query_nb, dim, k, dist, ind);
  }
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed_time, start, stop);
  printf(" done in %f s for %d iterations (%f s by iteration)\n",
      elapsed_time/1000, iterations, elapsed_time/(iterations*1000));

  // Destroy cuda event object and free memory
  hipEventDestroy(start);
  hipEventDestroy(stop);
  free(ind);
  free(dist);
  free(query);
  free(ref);
}

